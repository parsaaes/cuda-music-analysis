
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <hipfft/hipfft.h>
#include <stdlib.h>
#include <math.h>


void dft(int size, hipfftComplex * input, hipfftComplex * output) {
	hipfftHandle plan;
	hipfftComplex *d_input, *d_output;
	hipMalloc((void **)&d_input, size * sizeof(hipfftComplex));
	hipMalloc((void **)&d_output, size * sizeof(hipfftComplex));
	hipMemcpy(d_input, input, size * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	hipfftPlan1d(&plan, size, HIPFFT_C2C, 1);
	hipfftExecC2C(plan, d_input, d_output, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	hipDeviceSynchronize();
	hipMemcpy(output, d_output, size * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	hipFree(d_input);
	hipFree(d_output);
}

double* powerSpecterum(int size, hipfftComplex * wave) {
	double * result = (double*) malloc(size * sizeof(double));
	for (int i = 0; i < size; i++) {
		result[i] = wave[i].x * wave[i].x + wave[i].y * wave[i].y;
	}
	return result;
}

int main()
{
	hipSetDevice(0);
	hipfftComplex *input, *output;
	int size = 100;
	input = (hipfftComplex*)malloc(size * sizeof(hipfftComplex));
	output = (hipfftComplex*)malloc(size * sizeof(hipfftComplex));

	for (int i = 0; i < size; i++) {
		input[i].x = (float)cos(3.14 + 3.14 * i);
		input[i].y = 0;
	}

	dft(size, input, output);
	double* power = powerSpecterum(size, output);

	for (int i = 0; i < size; i++) {
		printf("%d- %f^2 + %f^2 = %f \n",i,output[i].x, output[i].y, power[i]);
	}

    return 0;
}
